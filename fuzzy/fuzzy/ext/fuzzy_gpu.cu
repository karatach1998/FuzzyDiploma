#include "hip/hip_runtime.h"
#include <stdio.h>
#include "fuzzy.h"

#define CACHE_LINE_SIZE 128
#define T_NUM 11
#define IMPL(a, b) (1 - (a) + (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define WARP_SIZE 32
#define WARP_MULTIPLE(x) (((x) + (WARP_SIZE-1)) & ~(WARP_SIZE-1))

#define CU_HANDLE_ERROR(e) cuda_handle_error((e), __FILE__, __LINE__, #e)

static void cuda_handle_error(hipError_t e, const char* file, int line, const char* src)
{
    if (e != hipSuccess)
    {
        fprintf(stderr, "CUDA-ERROR: %s:%d: %s <%s>\n", file, line, hipGetErrorString(e), src);
        exit(EXIT_FAILURE);
    }
}

__global__
void compute_ftp_kernel(const float** fsets_table, const unsigned* fsets_lens, const unsigned* fsets_dims,
                        const float** a0_table, const unsigned char* a_indices, float* ftp_buf,
                        unsigned N, unsigned n)
{
    unsigned attr_index = blockIdx.x;
    unsigned i, j, k;

    unsigned a_len = fsets_lens[attr_index];
    unsigned a_n = fsets_dims[attr_index];
    unsigned b_n = fsets_dims[n];
    unsigned ftp_buf_entry_sz = WARP_MULTIPLE(MAX(T_NUM, b_n));

    // __shared__ float a0_cache[a_n+1];
    // __shared__ float a_cache[a_len][a_n+1];
    // __shared__ float ftp[T_NUM];

    extern __shared__ float cache[];

    float* a0_cache = cache;
    float* a_cache = a0_cache + (a_n+1);
    float* ftp = a_cache + a_len * (a_n+1);

    for (i = threadIdx.x; i < a_n; i += blockDim.x) a0_cache[i] = a0_table[attr_index][i];
    if (threadIdx.x == 0) a0_cache[a_n] = a0_cache[a_n-1];
    for (k = 0; k < a_len; ++k) {
        for (i = threadIdx.x; i < a_n; i += blockDim.x) a_cache[k * (a_n+1) + i] = fsets_table[attr_index][k * a_n + i];
        a_cache[k * (a_n+1) + a_n] = a_cache[k * (a_n+1) + a_n - 1];
    }

    unsigned ti = threadIdx.x;
    float t = (float) ti / (T_NUM - 1);

    for (k = 0; k < N; ++k) {
        float* a = a_cache + a_indices[k + attr_index * N] * (a_n+1);

        ftp[ti] = 0.f;
        for (i = 0; i < a_n; ++i) {
            float a1 = a[i];
            float a2 = a[i + 1];

            if ((t - a1) * (a2 - t) >= 0) {
                float y = a0_cache[i] + (t - a1) * (a0_cache[i+1] - a0_cache[i]) / (a2 - a1);
                if (y > ftp[ti]) ftp[ti] = y;
            }
        }
        ftp_buf[(k * n + attr_index) * ftp_buf_entry_sz + ti] = ftp[ti];
    }
}

__global__
void compute_b0_kernel(const float** fsets_table, const unsigned* fsets_lens, const unsigned* fsets_dims,
                       const float* ftp_buf, const unsigned char* b_indices,
                       float* b0_buf, unsigned N, unsigned n)
{
    unsigned attr_index = blockIdx.x;
    unsigned i, ti, k;

    unsigned b_len = fsets_lens[n];
    unsigned b_n = fsets_dims[n];
    unsigned buf_entry_sz = WARP_MULTIPLE(MAX(T_NUM, b_n));

    extern __shared__ float cache[];

    float* b_cache = cache;
    float* b0 = b_cache + b_len * blockDim.x;

    for (k = 0; k < b_len; ++k) if (threadIdx.x < b_n) b_cache[k * blockDim.x + threadIdx.x] = fsets_table[n][k * b_n + threadIdx.x];

    for (k = 0; k < N; ++k) {
        float* b = b_cache + b_indices[k] * blockDim.x;

        b0[threadIdx.x] = 0.f;
        for (ti = 0; ti < T_NUM; ++ti) {
            float impl = IMPL((float)ti / (T_NUM - 1), b[threadIdx.x]);
            float min = MIN(ftp_buf[(k * n + attr_index) * buf_entry_sz + ti], impl);
            if (min > b0[threadIdx.x]) b0[threadIdx.x] = min;
        }
        __syncthreads();
        b0_buf[(k * n + attr_index) * buf_entry_sz + threadIdx.x] = b0[threadIdx.x];
    }
}

__global__
void reduce_kernel(const float* b0_buf, float* partial_b0, unsigned b_n, unsigned N, unsigned n)
{
    unsigned buf_entry_sz = WARP_MULTIPLE(MAX(T_NUM, b_n));
    unsigned warp_multiple_dim = WARP_MULTIPLE(b_n); // same as (blockDim.x / n)
    unsigned i = threadIdx.x % warp_multiple_dim;
    unsigned attr_index = threadIdx.x / warp_multiple_dim;
    unsigned k = blockIdx.x;
    unsigned step;

    extern __shared__ float buf[];
    float* b0_buf_cache = buf;
    float* b0_cache = b0_buf_cache + blockDim.x;

    if (attr_index == 0) b0_cache[i] = 1.f;
    while (k < N) {
        b0_buf_cache[threadIdx.x] = b0_buf[(k * n + attr_index) * buf_entry_sz + i];
        for (step = 1; step < n; step <<= 1) {
            if ((attr_index & ((step<<1)-1)) == 0 && attr_index + step < n) {
                b0_buf_cache[i + attr_index * warp_multiple_dim]
                    = fmaxf(b0_buf_cache[i + attr_index * warp_multiple_dim],
                            b0_buf_cache[i + (attr_index + step) * warp_multiple_dim]);
            }
            __syncthreads();
        }
        // if (attr_index == 0) {
        //     float max = 0.f;
        //     for (unsigned j = 0; j < 6; ++j) max = fmaxf(max, b0_buf_cache[i + j * warp_multiple_dim]);
        //     // b0_cache[i] = fminf(b0_cache[i], b0_buf_cache[i]);
        //     b0_cache[i] = fminf(b0_cache[i], max);
        // }
        if (attr_index == 0) {
            b0_cache[i] = fminf(b0_cache[i], b0_buf_cache[i]);
        }
        // NOTE(sergey): So we skip synchronization in this point of loop,
        // because for debug purpose we thoughtlessly assume that 0 <= i < 32
        // and therefore will be executed by single warp.
        k += gridDim.x;
    }
    // NOTE(sergey): Make sure that b0_buf and partial_b0 don't overlap.
    // In other case memory layout compression can currupt, theoretically yet unused, data of other blocks.
    if (attr_index == 0) partial_b0[i + blockIdx.x * warp_multiple_dim] = b0_cache[i];
}

static hipDeviceProp_t get_props_for_current_device()
{
    int device;
    hipDeviceProp_t prop;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    return prop;
}

extern "C"
void predict_gpu(const float** fsets[], const unsigned* fsets_lens, const unsigned* fsets_dims,
                 const float* a0_table[], const unsigned char* a_indices_table[],
                 const unsigned char* b_indices, float* b0, unsigned N, unsigned n)
{
    /*
     * GPU global memory layout:
     * +-----------------------------------+
     * | [META] Fuzzy sets lens + dims     |
     * +-----------------------------------+
     * | Fuzzy sets table (Ptrs to buffer) |
     * +-----------------------------------+
     * | Fuzzy sets buffer (data store)    |
     * +-----------------------------------+
     * | a0 table (Ptrs to buffer)         |
     * +-----------------------------------+
     * | a0 (data store)                   |
     * +-----------------------------------+
     * | a (indices in the F-sets table)   |
     * +-----------------------------------+
     * | b (indices in the F-sets table)   |
     * +-----------------------------------+
     * | Partial computs buffer (ftp | b0) |
     * | and final result  b0 (data store) |
     * +-----------------------------------+
     *
     * NOTE: N-th F-sets table entry corresponds to b's underling attribute.
     * NOTE: It's highly desirable all fset's dim values have low variance.
     * NOTE: Everywhere below '*_table' varialb ename means array of pointers.
     */



    static hipDeviceProp_t prop = get_props_for_current_device();

    unsigned i, j, k;
    unsigned fsets_buf_sz = 0;
    unsigned a0_buf_sz = 0;
    unsigned partial_buf_entry_sz = WARP_MULTIPLE(MAX(T_NUM, fsets_dims[n]));
    unsigned max_fsets_len = 0, max_fsets_dim = 0;

    for (i = 0; i < n + 1; ++i) fsets_buf_sz += fsets_lens[i] * fsets_dims[i];
    for (i = 0; i < n; ++i) a0_buf_sz += fsets_dims[i];
    for (i = 0; i < n; ++i) {
        if (fsets_lens[i] > max_fsets_len) max_fsets_len = fsets_lens[i];
        if (fsets_dims[i] > max_fsets_dim) max_fsets_dim = fsets_dims[i];
    }

    unsigned *fsets_lens_d, *fsets_dims_d;
    float *fsets_buf_d, **fsets_table_d, *fsets_table_tmp[n+1];
    float *a0_buf_d, **a0_table_d, *a0_table_tmp[n];
    unsigned char *a_indices_d, *b_indices_d;
    float *partial_buf_d;
    unsigned offset;

    // TODO(sergey): In the case we break GPU computation to three separate kernels,
    // we can overlap some host2device memory operations and kernel execution operations.
    CU_HANDLE_ERROR(hipMalloc(&fsets_lens_d, sizeof(unsigned[n+1])));
    CU_HANDLE_ERROR(hipMalloc(&fsets_dims_d, sizeof(unsigned[n+1])));
    CU_HANDLE_ERROR(hipMalloc(&fsets_table_d, sizeof(float*[n+1])));
    CU_HANDLE_ERROR(hipMalloc(&fsets_buf_d, sizeof(float[fsets_buf_sz])));
    CU_HANDLE_ERROR(hipMalloc(&a0_table_d, sizeof(float*[n])));
    CU_HANDLE_ERROR(hipMalloc(&a0_buf_d, sizeof(float[a0_buf_sz])));
    CU_HANDLE_ERROR(hipMalloc(&a_indices_d, sizeof(unsigned char[n * N])));
    CU_HANDLE_ERROR(hipMalloc(&b_indices_d, sizeof(unsigned char[N])));
    CU_HANDLE_ERROR(hipMalloc(&partial_buf_d, sizeof(float[partial_buf_entry_sz * n * N])));

    CU_HANDLE_ERROR(hipMemcpy(fsets_lens_d, fsets_lens, sizeof(unsigned[n+1]), hipMemcpyHostToDevice));
    CU_HANDLE_ERROR(hipMemcpy(fsets_dims_d, fsets_dims, sizeof(unsigned[n+1]), hipMemcpyHostToDevice));

    offset = 0;
    for (i = 0; i < n + 1; ++i) {
        fsets_table_tmp[i] = fsets_buf_d + offset;
        for (j = 0; j < fsets_lens[i]; ++j) CU_HANDLE_ERROR(hipMemcpy((fsets_buf_d + offset) + j * fsets_dims[i], fsets[i][j], sizeof(float[fsets_dims[i]]), hipMemcpyHostToDevice));
        offset += fsets_lens[i] * fsets_dims[i];
    }
    CU_HANDLE_ERROR(hipMemcpy(fsets_table_d, fsets_table_tmp, sizeof(fsets_table_tmp), hipMemcpyHostToDevice));

    offset = 0;
    for (i = 0; i < n; ++i) {
        a0_table_tmp[i] = a0_buf_d + offset;
        CU_HANDLE_ERROR(hipMemcpy(a0_buf_d + offset, a0_table[i], sizeof(float[fsets_dims[i]]), hipMemcpyHostToDevice));
        offset += fsets_dims[i];
    }
    CU_HANDLE_ERROR(hipMemcpy(a0_table_d, a0_table_tmp, sizeof(a0_table_tmp), hipMemcpyHostToDevice));

    for (i = 0; i < n; ++i) hipMemcpy(a_indices_d + i * N, a_indices_table[i], sizeof(unsigned char[N]), hipMemcpyHostToDevice);
    CU_HANDLE_ERROR(hipMemcpy(b_indices_d, b_indices, sizeof(unsigned char[N]), hipMemcpyHostToDevice));

    {
        unsigned blocks = n;
        unsigned warp_multiple_dim = WARP_MULTIPLE(T_NUM);
        unsigned threads = warp_multiple_dim;
        unsigned shared_sz = sizeof(float[/* a0_cache */ (max_fsets_dim+1) + /* a_cache */ max_fsets_len * (max_fsets_dim+1) + /* ftp */ warp_multiple_dim]);
        compute_ftp_kernel<<<blocks, threads, shared_sz>>>((const float**)fsets_table_d, fsets_lens_d, fsets_dims_d, (const float**)a0_table_d, a_indices_d, partial_buf_d, N, n);
        CU_HANDLE_ERROR(hipPeekAtLastError());
    }

    {
        unsigned blocks = n;
        unsigned warp_multiple_dim = WARP_MULTIPLE(fsets_dims[n]);
        unsigned threads = warp_multiple_dim;
        unsigned shared_sz = sizeof(float[/* b_cache + b0 */ (fsets_lens[n] + 1) * warp_multiple_dim]);
        compute_b0_kernel<<<blocks, threads, shared_sz>>>((const float**)fsets_table_d, fsets_lens_d, fsets_dims_d, partial_buf_d, b_indices_d, partial_buf_d, N, n);
        CU_HANDLE_ERROR(hipPeekAtLastError());
    }

    float* partial_b0_d;
    unsigned partial_b0_len, partial_b0_entry_sz;

    {
        float* b0_buf_d = partial_buf_d;

        // NOTE(sergey): Block count was taken from CUDA by Example book (Histogram computation with atomic operations),
        // where it was figured out experimentally that maximal performance is achieved,
        // when block number is exactly twice multiple of the number of multiprocessors.
        unsigned blocks = partial_b0_len = prop.multiProcessorCount * 2;
        unsigned warp_multiple_dim = partial_b0_entry_sz = WARP_MULTIPLE(fsets_dims[n]);
        unsigned threads = warp_multiple_dim * n;
        unsigned shared_sz = sizeof(float[n+1][warp_multiple_dim]);
        hipMalloc(&partial_b0_d, sizeof(float[partial_b0_len][partial_b0_entry_sz]));
        reduce_kernel<<<blocks, threads, shared_sz>>>(b0_buf_d, partial_b0_d, fsets_dims[n], N, n);
        CU_HANDLE_ERROR(hipPeekAtLastError());
    }

    float partial_b0[partial_b0_len][partial_b0_entry_sz];
    hipMemcpy(partial_b0, partial_b0_d, sizeof(float[partial_b0_len][partial_b0_entry_sz]), hipMemcpyDeviceToHost);
    // float partial_b0[N * n][partial_b0_entry_sz];
    // hipMemcpy(partial_b0, partial_buf_d, sizeof(float[N * n][partial_buf_entry_sz]), hipMemcpyDeviceToHost);

    memcpy(b0, partial_b0, sizeof(float[fsets_dims[n]]));
    for (i = 1; i < partial_b0_len; ++i) {
        for (j = 0; j < fsets_dims[n]; ++j) {
            // b0[j] = MIN(b0[j], partial_b0[i][j]);
            if (partial_b0[i][j] < b0[j]) b0[j] = partial_b0[i][j];
        }
    }

    // float b0_tmp[21];
    // for (j = 0; j < fsets_dims[n]; ++j) b0[j] = 1.f;
    // for (k = 0; k < N; ++k) {
    //     memset(b0_tmp, 0, sizeof(b0_tmp));
    //     for (i = 0; i < n; ++i) {
    //         for (j = 0; j < fsets_dims[n]; ++j) b0_tmp[j] = MAX(b0_tmp[j], partial_b0[i + k * n][j]);
    //     }
    //     for (j = 0; j < fsets_dims[n]; ++j) b0[j] = MIN(b0[j], b0_tmp[j]);
    // }

    hipFree(fsets_lens_d);
    hipFree(fsets_dims_d);
    hipFree(fsets_table_d);
    hipFree(fsets_buf_d);
    hipFree(a0_table_d);
    hipFree(a0_buf_d);
    hipFree(a_indices_d);
    hipFree(b_indices_d);
    hipFree(partial_buf_d);
    hipFree(partial_b0_d);
}
